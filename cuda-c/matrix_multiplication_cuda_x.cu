
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <vector>

// CUDA kernel to perform matrix multiplication
__global__ void matrixMultiplication(const int* M, const int* N, int* P, const int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < width * width) {
        int row = idx / width;
        int col = idx % width;

        int p_value = 0;
        for (int k = 0; k < width; ++k) {
            int m_value = M[row * width + k];
            int n_value = N[k * width + col];
            p_value += m_value * n_value;
        }
        P[idx] = p_value;
    }
}

int random_number() {
    return (std::rand() % 100);
}

void print_matrix(const int* M, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d", M[i * n + j]);
            if (j == n) continue;
            printf("\t");
        }
        printf("\n");
    }
}

int main(int argc, char** argv) {
    srand(time(NULL));

    const int n = 1024;  // Width of the matrices

    std::vector<int> A(n * n), B(n * n), C(n * n);  // Local variables, hosted in memory
    std::generate(A.begin(), A.end(), random_number);
    std::generate(B.begin(), B.end(), random_number);

    printf("Matrix A\n");
    //print_matrix(A.data(), n);

    printf("Matrix B\n");
    //print_matrix(B.data(), n);

    // Device matrices
    int* d_A;
    int* d_B;
    int* d_C;
    size_t matrixSize = n * n * sizeof(int);
    hipMalloc((void**)&d_A, matrixSize);
    hipMalloc((void**)&d_B, matrixSize);
    hipMalloc((void**)&d_C, matrixSize);

    // Copy host matrices to device
    hipMemcpy(d_A, A.data(), matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), matrixSize, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    int blockSize = 256;  // Adjust block size based on device capabilities
    int gridSize = (n * n + blockSize - 1) / blockSize;

    // Launch CUDA kernel
    matrixMultiplication<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);

    // Copy result matrix from device to host
    hipMemcpy(C.data(), d_C, matrixSize, hipMemcpyDeviceToHost);

    printf("Matrix C\n");
    //print_matrix(C.data(), n);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

