
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <vector>

__global__ void sum_kernel(const int *x,const int *y, int *res, int max_index){
    // All operands are passed by reference

    // The operation is to be executed on the device,
    // so the variables x,y,res must point to the memory of the GPU

    // The index refers to the innermost thread execution unit
    // thus the thread inside a block
    auto index = threadIdx.x + blockIdx.x * blockDim.x;

    // With arbitrary-sized vectors, we must protect the kernel 
    // from accessing a piece of memory of the device that has not been allocated
    if (index < max_index){
    	res[index] = x[index] + y[index];
    }
    printf("Block - Thread number %d-%d\n",blockIdx.x,threadIdx.x);
}

int random_number(){
    return (std::rand()%100);
}

int main(int argc, char **argv){

    srand(time(NULL));

    int N = 39;                         // Length of the vectors
    
    std::vector<int> a(N), b(N), c(N);  // Local variables, hosted in memory
    std::generate(a.begin(), a.end(), random_number);
    std::generate(b.begin(), b.end(), random_number);
    
    int *dev_a, *dev_b, *dev_c;         // Copies of the variables hosted on the device

    int size = N * sizeof(int);         // Size of the memory associated to each entry

    // Carve the memory on the device by allocating space 
    // for the copies hosted on the device
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, size);

    for (int i = 0; i < N; i++){
        printf("[el. %d] %d + %d = %d (on CPU) \n",i,a[i],b[i],a[i]+b[i]);
    }
    
    // Copy input variables to the device
    hipMemcpy(dev_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b.data(), size, hipMemcpyHostToDevice);

    // Launch the sum kernel on the GPU
    //  - 40 parallel executions of the kernel  
    //  --> 5 blocks of 8 threads each

    // We must be careful not to execute a kernel on a non-existing piece of data

    int N_threadsPerBlock = 10;
    sum_kernel<<<(N + N_threadsPerBlock - 1)/N_threadsPerBlock,N_threadsPerBlock>>>(dev_a, dev_b, dev_c, N);
    
    // Copy the result from the device back on the local variables
    hipMemcpy(c.data(), dev_c, size, hipMemcpyDeviceToHost);    
   
    for (int i = 0; i < N; i++){
        printf("[el. %d] %d + %d = %d (on GPU) \n",i,a[i],b[i],c[i]);
    }
   
    // Cleanup by freeing all used memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;

}
