
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <vector>

__global__ void sum_kernel(const int *x,const int *y, int *res){
    // All operands are passed by reference

    // The operation is to be executed on the device,
    // so the variables x,y,res must point to the memory of the GPU

    // threadIdx is the entity that provides the index
    // of the thread to which the kernel is assigned within a block
    res[threadIdx.x] = x[threadIdx.x] + y[threadIdx.x];

    printf("Thread number %d\n",threadIdx.x);
}

int random_number(){
    return (std::rand()%100);
}

int main(int argc, char **argv){

    srand(time(NULL));

    int N = 4;                          // Length of the vectors
    
    std::vector<int> a(N), b(N), c(N);  // Local variables, hosted in memory
    std::generate(a.begin(), a.end(), random_number);
    std::generate(b.begin(), b.end(), random_number);
    
    int *dev_a, *dev_b, *dev_c;         // Copies of the variables hosted on the device

    int size = N * sizeof(int);         // Size of the memory associated to each entry

    // Carve the memory on the device by allocating space 
    // for the copies hosted on the device
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, size);

    for (int i = 0; i < N; i++){
        printf("[el. %d] %d + %d = %d (on CPU) \n",i,a[i],b[i],a[i]+b[i]);
    }
    
    // Copy input variables to the device
    hipMemcpy(dev_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b.data(), size, hipMemcpyHostToDevice);

    // Launch the sum kernel on the GPU
    //  - 4 parallel executions of the kernel  
    sum_kernel<<<1,4>>>(dev_a, dev_b, dev_c);
    
    // Copy the result from the device back on the local variables
    hipMemcpy(c.data(), dev_c, size, hipMemcpyDeviceToHost);    
   
    for (int i = 0; i < N; i++){
        printf("[el. %d] %d + %d = %d (on GPU) \n",i,a[i],b[i],c[i]);
    }
   
    // Cleanup by freeing all used memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);        

    return 0;

}
